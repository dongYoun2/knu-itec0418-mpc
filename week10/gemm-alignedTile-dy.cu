#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
const float alpha = 0.5f;
const float beta = -100.0f;
const unsigned TILE_WIDTH = 32;
unsigned matsize = 4096; // num rows and also num cols

__global__ void kernelGEMM(float* matDst, float* matA, float* matB, float* matC, unsigned int matsize, int pitchInElem, const float alpha, const float beta) {
    unsigned int gy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float smatA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float smatB[TILE_WIDTH][TILE_WIDTH];

    int ntiles = matsize / TILE_WIDTH;
    float sum = 0;
    for (int tIdx=0; tIdx<ntiles; tIdx++) {
        unsigned int idxA = gy * pitchInElem + (tIdx * TILE_WIDTH + threadIdx.x);
        unsigned int idxB = (tIdx * TILE_WIDTH + threadIdx.y) * pitchInElem + gx;
        smatA[threadIdx.y][threadIdx.x] = matA[idxA];
        smatB[threadIdx.y][threadIdx.x] = matB[idxB];
        __syncthreads();

        for (int k=0; k<TILE_WIDTH; k++) {
            sum += smatA[threadIdx.y][k] * smatB[k][threadIdx.x];
        }
        __syncthreads();
    }
    unsigned int idx = gy * pitchInElem + gx;
    matDst[idx] = alpha * sum + beta * matC[idx];
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		matsize = procArg( argv[0], argv[1], 4 );
		break;
	default:
		printf("usage: %s [matsize]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}

    if (matsize % TILE_WIDTH != 0) {
		printf("%s: only accepts multiples of TILE_WIDTH which is %d\n", argv[0], TILE_WIDTH);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
	}
    
    float* matA = (float*)malloc(matsize * matsize * sizeof(float));
    float* matB = (float*)malloc(matsize * matsize * sizeof(float));
    float* matC = (float*)malloc(matsize * matsize * sizeof(float));
    float* matZ = (float*)malloc(matsize * matsize * sizeof(float));

    srand(0);
    setNormalizedRandomData(matA, matsize * matsize);
    setNormalizedRandomData(matB, matsize * matsize);
    setNormalizedRandomData(matC, matsize * matsize);

    float* dev_matA = nullptr;
    float* dev_matB = nullptr;
    float* dev_matC = nullptr;
    float* dev_matZ = nullptr;
    size_t dPitch = 0;

    ELAPSED_TIME_BEGIN(1);
    hipMallocPitch(&dev_matA, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matB, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matC, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matZ, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();

    size_t hostPitch = matsize * sizeof(float);
    hipMemcpy2D(dev_matA, dPitch, matA, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();
    hipMemcpy2D(dev_matB, dPitch, matB, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();
    hipMemcpy2D(dev_matC, dPitch, matC, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    dim3 dimBlock(32, 32);
    dim3 dimGrid(div_up(matsize, dimBlock.x), div_up(matsize, dimBlock.y));
    assert(dPitch % sizeof(float) == 0);
    int dPitchInElem = dPitch / sizeof(float);
    CUDA_PRINT_CONFIG_2D( matsize, matsize );
    ELAPSED_TIME_BEGIN(0);
    kernelGEMM<<<dimGrid, dimBlock>>>(dev_matZ, dev_matA, dev_matB, dev_matC, matsize, dPitchInElem, alpha, beta);
    hipDeviceSynchronize();
    ELAPSED_TIME_END(0);
    CUDA_CHECK_ERROR();
    
    hipMemcpy2D(matZ, hostPitch, dev_matZ, dPitch, matsize * sizeof(float), matsize, hipMemcpyDeviceToHost);
    ELAPSED_TIME_END(1);
    CUDA_CHECK_ERROR();
    
    hipFree(dev_matA);
    CUDA_CHECK_ERROR();
    hipFree(dev_matB);
    CUDA_CHECK_ERROR();
    hipFree(dev_matC);
    CUDA_CHECK_ERROR();
    hipFree(dev_matZ);
    CUDA_CHECK_ERROR();

    float sumA = getSum(matA, matsize * matsize);
    float sumB = getSum(matB, matsize * matsize);
    float sumC = getSum(matC, matsize * matsize);
    float sumZ = getSum(matZ, matsize * matsize);
    printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
    printf("sumA = %f\n", sumA);
    printf("sumB = %f\n", sumB);
    printf("sumC = %f\n", sumC);
    printf("sumZ = %f\n", sumZ);
    printMat("matZ", matZ, matsize, matsize);
    printMat("matA", matA, matsize, matsize);
    printMat("matB", matB, matsize, matsize);
    printMat("matC", matC, matsize, matsize);

    free(matA);
    free(matB);
    free(matC);
    free(matZ);

    return 0;
}