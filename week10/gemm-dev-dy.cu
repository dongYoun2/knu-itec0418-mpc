#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
const float alpha = 0.5f;
const float beta = -100.0f;
unsigned matsize = 4096; // num rows and also num cols

__global__ void kernelGEMM(float* matDst, float* matA, float* matB, float* matC, unsigned int matsize, int pitchInElem, const float alpha, const float beta) {
    unsigned int gy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gy < matsize && gx < matsize)  {
        unsigned idx = gy * pitchInElem + gx;
        float sum = 0;
        for (unsigned int k=0; k<matsize; k++) {
            unsigned idxA = gy * pitchInElem + k;
            unsigned idxB = k * pitchInElem + gx;
            sum += matA[idxA] * matB[idxB];
        }
        matDst[idx] = alpha * sum + beta * matC[idx];
    }
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		matsize = procArg( argv[0], argv[1], 4 );
		break;
	default:
		printf("usage: %s [matsize]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
    
    float* matA = (float*)malloc(matsize * matsize * sizeof(float));
    float* matB = (float*)malloc(matsize * matsize * sizeof(float));
    float* matC = (float*)malloc(matsize * matsize * sizeof(float));
    float* matZ = (float*)malloc(matsize * matsize * sizeof(float));

    srand(0);
    setNormalizedRandomData(matA, matsize * matsize);
    setNormalizedRandomData(matB, matsize * matsize);
    setNormalizedRandomData(matC, matsize * matsize);

    float* dev_matA = nullptr;
    float* dev_matB = nullptr;
    float* dev_matC = nullptr;
    float* dev_matZ = nullptr;
    size_t dPitch = 0;

    ELAPSED_TIME_BEGIN(1);
    hipMallocPitch(&dev_matA, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matB, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matC, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matZ, &dPitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();

    size_t hostPitch = matsize * sizeof(float);
    hipMemcpy2D(dev_matA, dPitch, matA, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();
    hipMemcpy2D(dev_matB, dPitch, matB, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();
    hipMemcpy2D(dev_matC, dPitch, matC, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    dim3 dimBlock(32, 32);
    dim3 dimGrid(div_up(matsize, dimBlock.x), div_up(matsize, dimBlock.y));
    assert(dPitch % sizeof(float) == 0);
    int dPitchInElem = dPitch / sizeof(float);
    CUDA_PRINT_CONFIG_2D( matsize, matsize );
    ELAPSED_TIME_BEGIN(0);
    kernelGEMM<<<dimGrid, dimBlock>>>(dev_matZ, dev_matA, dev_matB, dev_matC, matsize, dPitchInElem, alpha, beta);
    hipDeviceSynchronize();
    ELAPSED_TIME_END(0);
    CUDA_CHECK_ERROR();
    
    hipMemcpy2D(matZ, hostPitch, dev_matZ, dPitch, matsize * sizeof(float), matsize, hipMemcpyDeviceToHost);
    ELAPSED_TIME_END(1);
    CUDA_CHECK_ERROR();
    
    hipFree(dev_matA);
    CUDA_CHECK_ERROR();
    hipFree(dev_matB);
    CUDA_CHECK_ERROR();
    hipFree(dev_matC);
    CUDA_CHECK_ERROR();
    hipFree(dev_matZ);
    CUDA_CHECK_ERROR();

    float sumA = getSum(matA, matsize * matsize);
    float sumB = getSum(matB, matsize * matsize);
    float sumC = getSum(matC, matsize * matsize);
    float sumZ = getSum(matZ, matsize * matsize);
    printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
    printf("sumA = %f\n", sumA);
    printf("sumB = %f\n", sumB);
    printf("sumC = %f\n", sumC);
    printf("sumZ = %f\n", sumZ);
    printMat("matZ", matZ, matsize, matsize);
    printMat("matA", matA, matsize, matsize);
    printMat("matB", matB, matsize, matsize);
    printMat("matC", matC, matsize, matsize);

    free(matA);
    free(matB);
    free(matC);
    free(matZ);

    return 0;
}