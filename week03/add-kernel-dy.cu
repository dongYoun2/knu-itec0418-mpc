#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void printArray(const float* arr, int len, char name) {
    printf("%c: ", name);
    for (int i = 0; i < len; i++) {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

__global__ void addOne(float* dst, float* src) {
    int i = threadIdx.x;
    // 주의 1.0f 처럼 f 표시 해야함!
    dst[i] = src[i] + 1.0f;
}

int main(void) {
    // host-side data
	const int SIZE = 8;
	const float a[SIZE] = { 0., 1., 2., 3., 4., 5., 6., 7. };
	float b[SIZE] = { 0., 0., 0., 0., 0., 0., 0., 0. };

    printArray(a, SIZE, 'a');

    // device-side data
    float* dev_a = nullptr;
    float* dev_b = nullptr;

    hipMalloc((void**)&dev_a, SIZE * sizeof(float));
    hipMalloc((void**)&dev_b, SIZE * sizeof(float));

    hipMemcpy((void*)dev_a, a, SIZE * sizeof(float), hipMemcpyHostToDevice);

    addOne<<<1, SIZE>>>(dev_b, dev_a);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if(err != hipSuccess ) {
        printf("CUDA: ERROR: %s\n", hipGetErrorString(err));
        exit(1);
    } else {
        printf("CUDA: Success\n");
    }

    hipMemcpy((void*)b, dev_b, SIZE * sizeof(float), hipMemcpyDeviceToHost);
    printArray(b, SIZE, 'b');

    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}