#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
unsigned num = 16 * 1024 * 1024; // num data
const int MAX_THREAD_CNT = 1024;

__global__ void calcAdjDiff(float* dst, float* src, unsigned int n) {
    __shared__ float sharedMem[MAX_THREAD_CNT];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;
    if(i < n) {
        sharedMem[tx] = src[i];
        __syncthreads();
        
        if(tx > 0) {
            dst[i] = sharedMem[tx] - sharedMem[tx-1];
        } else if (i > 0 && tx == 0) {
            dst[i] = sharedMem[tx] - src[i-1];
        } else { // i == 0 (global index 가 0)
            dst[i] = sharedMem[tx] - 0.0F;
        }
    }
}

int main(const int argc, const char* argv[]) {
    // argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		num = procArg( argv[0], argv[1], 1 );
		break;
	default:
		printf("usage: %s [num]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}

    float* srcVec = (float*)malloc(num * sizeof(float));
    float* dstVec = (float*)malloc(num * sizeof(float));

    srand(0);
    setNormalizedRandomData(srcVec, num);
    
    float* dev_srcVec = nullptr;
    float* dev_dstVec = nullptr;

    ELAPSED_TIME_BEGIN(1);
    hipMalloc(&dev_srcVec, num * sizeof(float));
    CUDA_CHECK_ERROR();
    hipMalloc(&dev_dstVec, num * sizeof(float));
    CUDA_CHECK_ERROR();

    hipMemcpy(dev_srcVec, srcVec, num * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    dim3 dimBlock(MAX_THREAD_CNT);
    dim3 dimGrid(div_up(num, dimBlock.x));
    ELAPSED_TIME_BEGIN(0);
    calcAdjDiff<<<dimGrid, dimBlock>>>(dev_dstVec, dev_srcVec, num);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(0);

    hipMemcpy(dstVec, dev_dstVec, num * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(1);

    hipFree(dev_srcVec);
    CUDA_CHECK_ERROR();
    hipFree(dev_dstVec);
    CUDA_CHECK_ERROR();

    // check the result
	float sumA = getSum( srcVec, num );
	float sumB = getSum( dstVec, num );
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printVec( "srcVec", srcVec, num );
	printVec( "dstVec", dstVec, num );

    free(srcVec);
    free(dstVec);

    return 0;
}