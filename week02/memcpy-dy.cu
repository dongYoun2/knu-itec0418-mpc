#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


void printArray(const float* arr, int len) {
    for (int i = 0; i < len; i++) {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

int main(void) {
    // host-side data
    const int SIZE = 8;
    const float a[SIZE] = {1., 2., 3., 4., 5., 6., 7., 8.};
    float b[SIZE] = {0., 0., 0., 0., 0., 0., 0., 0.};

    printf("a: ");
    printArray(a, SIZE);

    // device-side data
    float* dev_ptr1 = nullptr;
    float* dev_ptr2 = nullptr;

    // 근데 이렇게 해도 동작은 함
    hipMalloc(&dev_ptr1, sizeof(a));
    hipMalloc(&dev_ptr2, sizeof(a));

    hipMemcpy(dev_ptr1, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(dev_ptr2, dev_ptr1, sizeof(a), hipMemcpyDeviceToDevice);
    hipMemcpy(b, dev_ptr2, sizeof(a), hipMemcpyDeviceToHost);

    // memory release 해주는 거 꼭 해줘야 함!
    hipFree(dev_ptr1);
    hipFree(dev_ptr2);

    printf("b: ");
    printArray(b, SIZE);

    return 0;
}