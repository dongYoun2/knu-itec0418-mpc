#include "hip/hip_runtime.h"
#include "./common.cpp"

unsigned matsize = 4000; // num rows and also num cols

int main(const int argc, const char* argv[]) {
    switch (argc) {
    case 1:
        break;
    case 2:
        matsize = procArg<int>(argv[0], argv[1], 4);
        break;
    default:
        printf("usage: %s [matsize]\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    float* matSrc = (float*)malloc(matsize * matsize * sizeof(float));
    float* matDst = (float*)malloc(matsize * matsize * sizeof(float));

    srand(0);
    setNormalizedRandomData(matSrc, matsize * matsize);

    float* dev_matSrc = nullptr;
    float* dev_matDst = nullptr;
    size_t dpitch = 0;

    ELAPSED_TIME_BEGIN(1);
    hipMallocPitch(&dev_matSrc, &dpitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();
    hipMallocPitch(&dev_matDst, &dpitch, matsize * sizeof(float), matsize);
    CUDA_CHECK_ERROR();

    size_t hostPitch = matsize * sizeof(float);
    hipMemcpy2D(dev_matSrc, dpitch, matSrc, hostPitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    dim3 dimBlock(32, 32);
    dim3 dimGrid(div_up(matsize, dimBlock.x), div_up(matsize, dimBlock.y));
    assert(dpitch % sizeof(float) == 0);
    ELAPSED_TIME_BEGIN(0);
    hipMemcpy2D(dev_matDst, dpitch, dev_matSrc, dpitch, sizeof(float) * matsize, matsize, hipMemcpyDeviceToDevice);
    // QUESTION: hipDeviceSynchronize() 호출 해야하나 말아야하나..
    // 안 하면 속도 훨씬 빠른데 synchronize() 해줘야하는 상황이 있는가..?
    // hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(0);

    hipMemcpy2D(matDst, hostPitch, dev_matDst, dpitch, matsize * sizeof(float), matsize, hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(1);

    hipFree(dev_matSrc);
    CUDA_CHECK_ERROR();
    hipFree(dev_matDst);
    CUDA_CHECK_ERROR();

    float sumSrc = getSum(matSrc, matsize * matsize);
    float sumDst = getSum(matDst, matsize * matsize);
    float diff = fabsf(sumSrc - sumDst);

    printf("matrix size: %d * %d\n", matsize, matsize);
    printf("sumSrc: %f\n", sumSrc);
    printf("sumDst: %f\n", sumDst);
    printf("diff(sumSrc, sumDst): %f\n", diff);
    printf("diff(sumSrc, sumDst) / SIZE: %f\n", diff / (matsize * matsize));
    printMat("Src", matSrc, matsize, matsize);
    printMat("Dst", matDst, matsize, matsize);

    free(matSrc);
    free(matDst);

    return 0;
}
